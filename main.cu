#include <iostream>
#include <iomanip>
#include <ctime>
#include <cfloat>
#include <pthread.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

using namespace std;

typedef double (*integrable)(double);

__host__ __device__ double parabola(double x)
{ return x * x; }

typedef struct integration_args_tag
{
    unsigned long long bstep;
    unsigned long long estep;
    double h;
    double a;
    double b;
    integrable f;
    double result;
} integration_args;



__global__ void integrate(double a, double b, integrable f, unsigned long long steps, double *result)
{
	extern __shared__ double pre_sums[];
	const unsigned long long thread_count = blockDim.x;
	unsigned long long thread_id = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned long long steps_per_thread = steps / thread_count;
    unsigned long long bstep = thread_id *  steps_per_thread;
    unsigned long long estep = bstep + steps_per_thread;
    pre_sums[thread_id] = 0;
    double h = (b - a) / (steps * 1.0);

    for (unsigned long long i = bstep; i < estep; i++)
    {
        double x = a + i * h;
        pre_sums[thread_id] += h * f(x);
    }

    __syncthreads();
    if(thread_id == 0)
    {
    	*result = 0;
    	for (unsigned long long i = 0; i < thread_count; i++) *result += pre_sums[i];
    }

}

__device__ integrable d_f = parabola;


int main(int argc, char *argv[])
{
    const unsigned int thread_count = 512;
    const unsigned long long steps = 1000000000;
    double a = 0;
    double b = 1;
    unsigned long long steps_per_thread = steps / thread_count;
    double *d_result;
    hipMalloc(&d_result, sizeof(double));
    integrable h_fun;
    hipMemcpyFromSymbol(&h_fun, HIP_SYMBOL(d_f), sizeof(integrable));
    hipEvent_t begin, end;
    hipEventCreate(&begin);
    hipEventCreate(&end);
    hipEventRecord(begin);
    int total_blocks = 4;
    dim3 grid;
    grid.x = total_blocks;
    dim3 block_geom;
    block_geom.x = thread_count / total_blocks;
    integrate<<<grid, block_geom,  sizeof(double) * thread_count>>>(a, b, h_fun, steps, d_result);
    hipEventRecord(end);
    hipEventSynchronize(end);
    double sum;
    hipMemcpy(&sum, d_result, sizeof(double), hipMemcpyDeviceToHost);
    cout << fixed << setprecision(DBL_DIG) << sum << endl;
    float time;
    hipEventElapsedTime(&time, begin, end);
    cout << fixed << setprecision(DBL_DIG) << time / 1000  << " seconds." << endl;
    hipFree(d_result);
    hipEventDestroy(begin);
    hipEventDestroy(end);
    return 0;
}
